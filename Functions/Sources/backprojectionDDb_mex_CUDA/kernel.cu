#include "hip/hip_runtime.h"
/*
%% Author: Rodrigo de Barros Vimieiro
% Date: March, 2019
% rodrigo.vimieiro@gmail.com
% =========================================================================
%{
% -------------------------------------------------------------------------
%                 backprojectionDDb_mex_CUDA(proj,param)
% -------------------------------------------------------------------------
%     DESCRIPTION:
%     This function reconstruct the 3D volume from projections, based on
%     the Distance-Driven principle. It works by calculating the overlap
%     in X and Y axis of the volume and the detector boundaries.
%     The geometry is for DBT with half cone-beam. All parameters are set
%     in "ParameterSettings" code.
%
%     INPUT:
%
%     - proj = 2D projections for each angle
%     - param = Parameter of all geometry
%	  - nProj = projection number to be backprojected
%
%     OUTPUT:
%
%     - data3d = reconstructed volume.
%
%     Reference:
%     - Branchless Distance Driven Projection and Backprojection,
%     Samit Basu and Bruno De Man (2006)
%     - GPU Acceleration of Branchless Distance Driven Projection and
%     Backprojection, Liu et al (2016)
%     - GPU-Based Branchless Distance-Driven Projection and Backprojection,
%     Liu et al (2017)
%     - A GPU Implementation of Distance-Driven Computed Tomography,
%     Ryan D. Wagner (2017)
%     ---------------------------------------------------------------------
%     Copyright (C) <2019>  <Rodrigo de Barros Vimieiro>
%
%     This program is free software: you can redistribute it and/or modify
%     it under the terms of the GNU General Public License as published by
%     the Free Software Foundation, either version 3 of the License, or
%     (at your option) any later version.
%
%     This program is distributed in the hope that it will be useful,
%     but WITHOUT ANY WARRANTY; without even the implied warranty of
%     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
%     GNU General Public License for more details.
%
%     You should have received a copy of the GNU General Public License
%     along with this program.  If not, see <http://www.gnu.org/licenses/>.
%}
% =========================================================================
%% 3-D Distance Driven Back-projection Code
*/

#include "backprojectionDDb_mex_cuda.h"


/****************************************************************************
*							CUDA Kernels									*
****************************************************************************/
__global__ void pad_projections_kernel(double* d_img, const int nDetXMap, const int nDetYMap, const int nElem, unsigned int np) {

	const int threadGId = blockIdx.x * blockDim.x + threadIdx.x;

	// Make sure we don't try and access memory outside
	// by having any threads mapped there return early
	if (threadGId >= nElem)
		return;

	d_img[(np*nDetYMap *nDetXMap) + (threadGId*nDetYMap)] = 0;

	return;
}

__global__ void map_boudaries_kernel(double* d_pBound, const int nElem, const double valueLeftBound, const double sizeElem, const double offset) {

	const int threadGId = blockIdx.x * blockDim.x + threadIdx.x;

	// Make sure we don't try and access memory outside
	// by having any threads mapped there return early
	if (threadGId >= nElem)
		return;

	d_pBound[threadGId] = (threadGId - valueLeftBound) * sizeElem + offset;

	return;
}

__global__ void rot_detector_kernel(double* d_pRdetY, double* d_pRdetZ, double* d_pYcoord, double* d_pZcoord, const double yOffset, const double zOffset,
	const double phi, const int nElem) {

	const int threadGId = blockIdx.x * blockDim.x + threadIdx.x;

	// Make sure we don't try and access memory outside
	// by having any threads mapped there return early
	if (threadGId >= nElem)
		return;

	// cos and sin are in measured in radians.

	d_pRdetY[threadGId] = ((d_pYcoord[threadGId] - yOffset)* cos(phi) - (d_pZcoord[threadGId] - zOffset)* sin(phi)) + yOffset;
	d_pRdetZ[threadGId] = ((d_pYcoord[threadGId] - yOffset)* sin(phi) + (d_pZcoord[threadGId] - zOffset)* cos(phi)) + zOffset;

	return;

}

__global__ void mapDet2Slice_kernel(double* const pXmapp, double* const pYmapp, double tubeX, double tubeY, double tubeZ, double* const pXcoord,
	double* const pYcoord, double* const pZcoord, double* const pZSlicecoord, const int nDetXMap, const int nDetYMap, const unsigned int nz) {

	/*

	Note: Matlab has linear indexing as a column of elements, i.e, the elements are actually stored in memory as queued columns.
	So threads in X are launched in the column direction (DBT Y coord), and threads in Y are launched in the row direction (DBT X coord).

	*/

	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	const int thread_1D_pos = thread_2D_pos.y * nDetYMap + thread_2D_pos.x;

	// Make sure we don't try and access memory outside the detector
	// by having any threads mapped there return early
	if (thread_2D_pos.x >= nDetYMap || thread_2D_pos.y >= nDetXMap)
		return;

	pXmapp[thread_1D_pos] = ((pXcoord[thread_2D_pos.y] - tubeX)*(pZSlicecoord[nz] - pZcoord[thread_2D_pos.x]) - (pXcoord[thread_2D_pos.y] * tubeZ) + (pXcoord[thread_2D_pos.y] * pZcoord[thread_2D_pos.x])) / (-tubeZ + pZcoord[thread_2D_pos.x]);

	if (thread_2D_pos.y == 0)
		pYmapp[thread_2D_pos.x] = ((pYcoord[thread_2D_pos.x] - tubeY)*(pZSlicecoord[nz] - pZcoord[thread_2D_pos.x]) - (pYcoord[thread_2D_pos.x] * tubeZ) + (pYcoord[thread_2D_pos.x] * pZcoord[thread_2D_pos.x])) / (-tubeZ + pZcoord[thread_2D_pos.x]);

	return;
}

__global__ void img_integration_kernel(double* d_img, const int nPixX, const int nPixY, bool direction, unsigned int offsetX, unsigned int offsetY, unsigned int nSlices) {

	/*

	Integration of 2D slices over the whole volume

	(S.1.Integration. - Liu et al(2017))

	** Perfom an inclusive scan **

	*/

	const int3 memory_2D_pos = make_int3(blockIdx.x * blockDim.x + threadIdx.x + offsetX,
		blockIdx.y * blockDim.y + threadIdx.y + offsetY,
		blockIdx.z * blockDim.z + threadIdx.z);

	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);


	// Make sure we don't try and access memory outside the detector
	// by having any threads mapped there return early
	if (memory_2D_pos.x >= nPixY || memory_2D_pos.y >= nPixX || memory_2D_pos.z >= nSlices)
		return;


	if (direction == integrateXcoord) {

		for (int s = 1; s <= blockDim.y; s *= 2) {

			int spot = thread_2D_pos.y - s;

			double val = 0;

			if (spot >= 0) {
				val = d_img[(memory_2D_pos.z*nPixY*nPixX) + (offsetY + spot) * nPixY + memory_2D_pos.x];
			}
			__syncthreads();

			if (spot >= 0) {
				d_img[(memory_2D_pos.z*nPixY*nPixX) + (memory_2D_pos.y * nPixY) + memory_2D_pos.x] += val;
			}
			__syncthreads();
		}
	}
	else
	{

		for (int s = 1; s <= blockDim.x; s *= 2) {

			int spot = thread_2D_pos.x - s;

			double val = 0;

			if (spot >= 0) {
				val = d_img[(memory_2D_pos.z*nPixY*nPixX) + memory_2D_pos.y * nPixY + spot + offsetX];
			}
			__syncthreads();

			if (spot >= 0) {
				d_img[(memory_2D_pos.z*nPixY*nPixX) + (memory_2D_pos.y * nPixY) + memory_2D_pos.x] += val;
			}
			__syncthreads();
		}

	}
	return;
}

//__global__ void bilinear_interp_kernel(double* d_interpData, hipTextureObject_t texObj, double* const d_pDetmX, double* const d_pDetmY, const int nDetXMap, const int nDetYMap, const double2 normFactor, const double2 offset)
//{
//
//	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
//										 blockIdx.y * blockDim.y + threadIdx.y);
//
//	// Make sure we don't try and access memory outside the detector
//	// by having any threads mapped there return early
//	if (thread_2D_pos.x >= nDetYMap || thread_2D_pos.y >= nDetXMap)
//		return;
//
//	// Normalize coordinates from 0-1 (coords on texture memory)
//	double xQuery = d_pDetmX[thread_2D_pos.y * nDetYMap + thread_2D_pos.x] / normFactor.x + offset.x + (-.5f * (1 / normFactor.x));
//	double yQuery = d_pDetmY[thread_2D_pos.x] / normFactor.y + offset.y + (.5f * (1 / normFactor.y));
//
//	// Read from texture and write to global memory
//	d_interpData[thread_2D_pos.y * nDetYMap + thread_2D_pos.x] = tex2D<double>(texObj, xQuery, yQuery);
//
//	return;
//}

__global__ void bilinear_interpolation_kernel_GPU(double* d_sliceI, double* d_pProj, double* d_pObjX, double* d_pObjY, double* d_pDetmX, double* d_pDetmY, const int nPixXMap, const int nPixYMap,
	const int nDetXMap, const int nDetYMap, const int nDetX, const int nDetY, const unsigned int np) {

	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
										 blockIdx.y * blockDim.y + threadIdx.y);

	// Make sure we don't try and access memory outside the detector
	// by having any threads mapped there return early
	if (thread_2D_pos.x >= nPixYMap || thread_2D_pos.y >= nPixXMap)
		return;

	/*

	S.2. Interpolation - Liu et al (2017)

	Reference:
	- https://en.wikipedia.org/wiki/Bilinear_interpolation
	- https://stackoverflow.com/questions/21128731/bilinear-interpolation-in-c-c-and-cuda

	Note: *** We are using the Unit Square equation ***

	alpha = X - X1
	1-alpha = X2 - X

	beta = Y - Y1
	1-beta = Y2 - Y

	----> Xcoord (thread_2D_pos.y)
	|							0___________
	v							|_d00_|_d10_|
	Ycoord (thread_2D_pos.x)	|_d01_|_d11_|

	Matlab code --
	objX = nDetX - (objX ./ detmX(1, end - 1));
	objY = (objY ./ detmX(1, end - 1)) - (detmY(1) / detmX(1, end - 1));

	*/

	// Adjust the mapped coordinates to cross the range of (0-nDetX).*duMap 
	//  Divide by pixelSize to get a unitary pixel size
	const double xNormData = nDetX - d_pObjX[thread_2D_pos.y] / d_pDetmX[0];
	const signed int    xData = floor(xNormData);
	const double  alpha = xNormData - xData;

	// Adjust the mapped coordinates to cross the range of (0-nDetY).*dyMap  
	//  Divide by pixelSize to get a unitary pixel size
	const double yNormData = (d_pObjY[thread_2D_pos.x] / d_pDetmX[0]) - (d_pDetmY[0] / d_pDetmX[0]);
	const signed int    yData = floor(yNormData);
	const double  beta = yNormData - yData;

	double d00, d01, d10, d11;
	if (((xNormData) >= 0) && ((xNormData) <= nDetX) && ((yNormData) >= 0) && ((yNormData) <= nDetY))	d00 = d_pProj[(np*nDetYMap*nDetXMap) + (xData*nDetYMap + yData)];						else    d00 = 0.0;
	if (((xData + 1) > 0) && ((xData + 1) <= nDetX) && ((yNormData) >= 0) && ((yNormData) <= nDetY))	d10 = d_pProj[(np*nDetYMap*nDetXMap) + ((xData + 1)*nDetYMap + yData)]; 				else    d10 = 0.0;
	if (((xNormData) >= 0) && ((xNormData) <= nDetX) && ((yData + 1) > 0) && ((yData + 1) <= nDetY))	d01 = d_pProj[(np*nDetYMap*nDetXMap) + (xData*nDetYMap + yData + 1)]; 				else    d01 = 0.0;
	if (((xData + 1) > 0) && ((xData + 1) <= nDetX) && ((yData + 1) > 0) && ((yData + 1) <= nDetY))		d11 = d_pProj[(np*nDetYMap*nDetXMap) + ((xData + 1)*nDetYMap + yData + 1)];			else    d11 = 0.0;

	double result_temp1 = alpha * d10 + (-d00 * alpha + d00);
	double result_temp2 = alpha * d11 + (-d01 * alpha + d01);

	d_sliceI[thread_2D_pos.y * nPixYMap + thread_2D_pos.x] = beta * result_temp2 + (-result_temp1 * beta + result_temp1);

}

__global__ void differentiation_kernel(double* d_pVolume, double* d_sliceI, double tubeX, double rtubeY, double rtubeZ,
	double* const d_pObjX, double* const d_pObjY, double* const d_pObjZ, const int nPixX, const int nPixY, const int nPixXMap,
	const int nPixYMap, const double du, const double dv, const double dx, const double dy, const double dz, const unsigned int nz) {

	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	const int thread_1D_pos = (nPixX*nPixY*nz) + (thread_2D_pos.y * nPixY) + thread_2D_pos.x;

	// Make sure we don't try and access memory outside the detector
	// by having any threads mapped there return early
	if (thread_2D_pos.x >= nPixY || thread_2D_pos.y >= nPixX)
		return;

	/*

	S.3. Differentiation - Eq. 24 - Liu et al (2017)

	Detector integral projection
	___________
	|_A_|_B_|___|
	|_C_|_D_|___|
	|___|___|___|


	(thread_2D_pos.x,thread_2D_pos.y)
	________________
	|_A_|__B__|_____|
	|_C_|(0,0)|(0,1)|
	|___|(1,0)|(1,1)|

	Threads are lauched from D up to nPixX (thread_2D_pos.y) and nPixY (thread_2D_pos.x)
	i.e., they are running on the detector image. Thread (0,0) is on D.

	Coordinates on intergal projection:

	A = thread_2D_pos.y * nPixYMap + thread_2D_pos.x
	B = ((thread_2D_pos.y+1) * nPixYMap) + thread_2D_pos.x
	C = thread_2D_pos.y * nPixYMap + thread_2D_pos.x + 1
	D = ((thread_2D_pos.y+1) * nPixYMap) + thread_2D_pos.x + 1

	*/

	unsigned int coordA = thread_2D_pos.y * nPixYMap + thread_2D_pos.x;
	unsigned int coordB = ((thread_2D_pos.y + 1) * nPixYMap) + thread_2D_pos.x;
	unsigned int coordC = coordA + 1;
	unsigned int coordD = coordB + 1;

	// x - ray angle in X coord
	double gamma = atan((d_pObjX[thread_2D_pos.y] + (dx / 2.0) - tubeX) / (rtubeZ - d_pObjZ[nz]));

	// x - ray angle in Y coord
	double alpha = atan((d_pObjY[thread_2D_pos.x] + (dy / 2.0) - rtubeY) / (rtubeZ - d_pObjZ[nz]));


	double dA, dB, dC, dD;

	dA = d_sliceI[coordA];
	dB = d_sliceI[coordB];
	dC = d_sliceI[coordC];
	dD = d_sliceI[coordD];

	// Treat border of interpolated integral detector
	if (dC == 0 && dD == 0) {
		dC = dA;
		dD = dB;
	}


	// S.3.Differentiation - Eq. 24 - Liu et al(2017)
	d_pVolume[thread_1D_pos] += ((dD - dC - dB + dA)*(du*dv*dz / (cos(alpha)*cos(gamma)*dx*dy)));

	return;
}

__global__ void division_kernel(double* d_img, const int nPixX, const int nPixY, unsigned int nSlices, unsigned int nProj){

	const int3 thread_2D_pos = make_int3(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y,
		blockIdx.z * blockDim.z + threadIdx.z);

	const int thread_1D_pos = (nPixX*nPixY*thread_2D_pos.z) + (thread_2D_pos.y * nPixY) + thread_2D_pos.x;

	// Make sure we don't try and access memory outside the detector
	// by having any threads mapped there return early
	if (thread_2D_pos.x >= nPixY || thread_2D_pos.y >= nPixX || thread_2D_pos.z >= nSlices)
		return;

	d_img[thread_1D_pos] /= (double) nProj;

}


/****************************************************************************
* function: backprojectionDDb() - CUDA backprojection Branchless Distance Driven.	*
****************************************************************************/

void backprojectionDDb(double* const h_pVolume,
	double* const h_pProj,
	double* const h_pTubeAngle,
	double* const h_pDetAngle,
	const double idXProj,
	const unsigned int nProj,
	const unsigned int nPixX,
	const unsigned int nPixY,
	const unsigned int nSlices,
	const unsigned int nDetX,
	const unsigned int nDetY,
	const double dx,
	const double dy,
	const double dz,
	const double du,
	const double dv,
	const double DSD,
	const double DDR,
	const double DAG)
{

	// Unique GPU
	int devID = 0;

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, devID);
	cudaCheckErrors("hipGetDeviceProperties");

	const unsigned int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;


	mexPrintf("GPU Device %d: \"%s\" with compute capability %d.%d has %d Multi-Processors and %zu bytes of global memory\n\n", devID,
		deviceProp.name, deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount, deviceProp.totalGlobalMem);

	
	// Create timmer variables and start it
	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);
	
	//hipStream_t stream1;
	//hipStreamCreate(&stream1);

	dim3 threadsPerBlock(1, 1, 1);
	dim3 blockSize(1, 1, 1);


	// Number of mapped detector and pixels
	const int nDetXMap = nDetX + 1;
	const int nDetYMap = nDetY + 1;
	const int nPixXMap = nPixX + 1;
	const int nPixYMap = nPixY + 1;


	// Convention: h_ variables live on host
	// Convention: d_ variables live on device (GPU global mem)
	double* d_pProj;
	double* d_sliceI;
	double* d_pVolume;
	double* d_pTubeAngle;
	double* d_pDetAngle;
	
	
	// Allocate global memory on the device, place result in "d_----"
	hipMalloc((void **)&d_pProj, nDetXMap*nDetYMap*nProj * sizeof(double)); 
	hipMalloc((void **)&d_sliceI, nPixXMap*nPixYMap * sizeof(double));
	hipMalloc((void **)&d_pVolume, nPixX*nPixY*nSlices * sizeof(double));
	hipMalloc((void **)&d_pTubeAngle, nProj * sizeof(double));
	hipMalloc((void **)&d_pDetAngle, nProj * sizeof(double));
	
	cudaCheckErrors("hipMalloc Initial");
	
	// Copy data from host memory "h_----" to device memory "d_----"
	hipMemcpy((void *)d_pTubeAngle, (void *)h_pTubeAngle, nProj * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy((void *)d_pDetAngle, (void *)h_pDetAngle, nProj * sizeof(double), hipMemcpyHostToDevice);

	cudaCheckErrors("hipMemcpy Angles");
	

	/*
	Copy projection data from host memory "h_----" to device memory "d_----" padding with zeros for image integation
	*/
	

	// Initialize first column and row with zeros
	double* h_pProj_tmp;
	double* d_pProj_tmp;

	threadsPerBlock.x = maxThreadsPerBlock;
	blockSize.x = (nDetXMap / maxThreadsPerBlock) + 1;

	for (unsigned int np = 0; np < nProj; np++) {

		// Pad on X coord direction
		pad_projections_kernel << <blockSize, threadsPerBlock >> > (d_pProj, nDetXMap, nDetYMap, nDetXMap, np);

		// Pad on Y coord direction
		d_pProj_tmp = d_pProj + (nDetXMap*nDetYMap*np) + 1;
		hipMemset(d_pProj_tmp, 0, nPixY * sizeof(double));
		cudaCheckErrors("hipMemset Padding Projections");
	}
	
	
	// Copy projections data from host memory
	for (unsigned int np = 0; np < nProj; np++)
		for (unsigned int c = 0; c < nDetX; c++) {

			h_pProj_tmp = h_pProj + (c *nDetY) + (nDetX*nDetY*np);
			d_pProj_tmp = d_pProj + (((c + 1) *nDetYMap) + 1) + (nDetXMap*nDetYMap*np);

			hipMemcpy((void *)d_pProj_tmp, (void *)h_pProj_tmp, nDetY * sizeof(double), hipMemcpyHostToDevice);
			cudaCheckErrors("hipMemcpy Projections");
		}

	
	
	// Pointer for projections coordinates
	double* d_pDetX;
	double* d_pDetY;
	double* d_pDetZ;
	double* d_pObjX;
	double* d_pObjY;
	double* d_pObjZ;

	// Allocate global memory on the device for projections coordinates
	hipMalloc((void **)&d_pDetX, nDetXMap * sizeof(double));
	hipMalloc((void **)&d_pDetY, nDetYMap * sizeof(double));
	hipMalloc((void **)&d_pDetZ, nDetYMap * sizeof(double));
	hipMalloc((void **)&d_pObjX, nPixXMap * sizeof(double));
	hipMalloc((void **)&d_pObjY, nPixYMap * sizeof(double));
	hipMalloc((void **)&d_pObjZ, nSlices * sizeof(double));

	cudaCheckErrors("hipMalloc Coordinates");



	// Pointer for mapped coordinates
	double* d_pDetmY;
	double* d_pDetmX;


	// Allocate global memory on the device for mapped coordinates
	hipMalloc((void **)&d_pDetmY, nDetYMap * sizeof(double));
	hipMalloc((void **)&d_pDetmX, nDetYMap * nDetXMap * sizeof(double));

	cudaCheckErrors("hipMalloc Map-Coordinates");


	// Pointer for rotated detector coords
	double* d_pRdetY;
	double* d_pRdetZ;

	// Allocate global memory on the device for for rotated detector coords
	hipMalloc((void **)&d_pRdetY, nDetYMap * sizeof(double));
	hipMalloc((void **)&d_pRdetZ, nDetYMap * sizeof(double));

	cudaCheckErrors("hipMalloc Rot-Coordinates");
	
	// Generate detector and object boudaries
	
	threadsPerBlock.x = maxThreadsPerBlock;

	blockSize.x = (nDetX / maxThreadsPerBlock) + 1;

	map_boudaries_kernel << <blockSize, threadsPerBlock >> > (d_pDetX, nDetXMap, (double)nDetX, -du, 0.0);

	blockSize.x = (nDetY / maxThreadsPerBlock) + 1;

	map_boudaries_kernel << <blockSize, threadsPerBlock >> > (d_pDetY, nDetYMap, nDetY / 2.0, dv, 0.0);

	blockSize.x = (nPixX / maxThreadsPerBlock) + 1;

	map_boudaries_kernel << <blockSize, threadsPerBlock >> > (d_pObjX, nPixXMap, (double)nPixX, -dx, 0.0);

	blockSize.x = (nPixY / maxThreadsPerBlock) + 1;

	map_boudaries_kernel << <blockSize, threadsPerBlock >> > (d_pObjY, nPixYMap, nPixY / 2.0, dy, 0.0);

	blockSize.x = (nSlices / maxThreadsPerBlock) + 1;

	map_boudaries_kernel << <blockSize, threadsPerBlock >> > (d_pObjZ, nSlices, 0.0, dz, DAG + (dz / 2.0));

	
	//mexPrintf("Map Boundaries -- Threads:%d Blocks:%d \n", threads, blocks);

	
	// Initiate variables value with 0
	hipMemset(d_pDetZ, 0, nDetYMap * sizeof(double));
	hipMemset(d_pVolume, 0, nPixX * nPixY * nSlices * sizeof(double));

	cudaCheckErrors("hipMemset Zeros");
	
	//hipDeviceSynchronize();

	// X - ray tube initial position
	double tubeX = 0;
	double tubeY = 0;
	double tubeZ = DSD;

	// Iso - center position
	double isoY = 0;
	double isoZ = DDR;
	

	// Integration of 2D projection over the whole projections
	// (S.1.Integration. - Liu et al(2017))

	// Naive integration o the X coord
	threadsPerBlock.x = 10;
	threadsPerBlock.y = 10;
	threadsPerBlock.z = 10;

	blockSize.x = (unsigned int)ceil(((double)nDetYMap / (threadsPerBlock.x - 1)));
	blockSize.y = 1;
	blockSize.z = (unsigned int)ceil((double)nProj / threadsPerBlock.z);

	//mexPrintf("Divisao: %f \n", (double)nDetY / (threadsPerBlock.x - 1));
	//mexPrintf("Divisao ceil: %d \n", (unsigned int)ceil((double)nDetY / (threadsPerBlock.x - 1)));

	for (int k = 0; k < ceil((double)nDetXMap / (threadsPerBlock.x - 1)); k++) {

		img_integration_kernel << <blockSize, threadsPerBlock >> > (d_pProj, nDetXMap, nDetYMap, integrateXcoord, 0, k * 9, nProj);

		//mexPrintf("integration kernel -- threadsX:%d blocksX:%d threadsY:%d blocksY:%d \n", threadsPerBlock.x, blockSize.x, threadsPerBlock.y, blockSize.y);

	}

	// Naive integration o the Y coord
	threadsPerBlock.x = 10;
	threadsPerBlock.y = 10;
	threadsPerBlock.z = 10;

	blockSize.x = 1;
	blockSize.y = (unsigned int)ceil((double)nDetXMap / (threadsPerBlock.y - 1));
	blockSize.z = (unsigned int)ceil((double)nProj / threadsPerBlock.z);


	for (int k = 0; k < ceil((double)nDetYMap / (threadsPerBlock.y - 1)); k++) {

		img_integration_kernel << <blockSize, threadsPerBlock >> > (d_pProj, nDetXMap, nDetYMap, integrateYcoord, k * 9, 0, nProj);

		//mexPrintf("integration kernel -- threadsX:%d blocksX:%d threadsY:%d blocksY:%d \n", threadsPerBlock.x, blockSize.x, threadsPerBlock.y, blockSize.y);

	}
	
	double* d_pDetmX_tmp = d_pDetmX + (nDetYMap * (nDetXMap-2));

	// Test if we will loop over all projs or not
	unsigned int projIni, projEnd, nProj2Run;
	if(idXProj == -1){
		projIni = 0;
		projEnd = nProj;
		nProj2Run = nProj;
	}
	else{
		nProj2Run = 1;
		projIni = (unsigned int) idXProj;
		projEnd = (unsigned int) idXProj + 1;
	}
	
	
	// For each projection
	for (unsigned int p = projIni; p < projEnd; p++) {
	
		// Get specif tube angle for the projection
		double theta = h_pTubeAngle[p] * M_PI / 180.0;

		// Get specif detector angle for the projection
		double phi = h_pDetAngle[p] * M_PI / 180.0;

		//mexPrintf("Tube angle:%f Det angle:%f\n", theta, phi);

		// Tube rotation
		double rtubeY = ((tubeY - isoY)*cos(theta) - (tubeZ - isoZ)*sin(theta)) + isoY;
		double rtubeZ = ((tubeY - isoY)*sin(theta) + (tubeZ - isoZ)*cos(theta)) + isoZ;

		//mexPrintf("R tube Y:%f R tube Z:%f\n", rtubeY, rtubeZ);

		// Detector rotation
		threadsPerBlock.x = maxThreadsPerBlock;
		threadsPerBlock.y = 1;
		threadsPerBlock.z = 1;

		blockSize.x = (nDetYMap / maxThreadsPerBlock) + 1;
		blockSize.y = 1;
		blockSize.z = 1;

		rot_detector_kernel << <blockSize, threadsPerBlock >> > (d_pRdetY, d_pRdetZ, d_pDetY, d_pDetZ, isoY, isoZ, phi, nDetYMap);

		//hipDeviceSynchronize();

		//mexPrintf("Detector rotation -- Threads:%d Blocks:%d \n", threads, blocks);


		// For each slice
		for (unsigned int nz = 0; nz < nSlices; nz++) {
			
			/*

			Map detector onto XY plane(Inside proj loop in case detector rotates)

			*** Note: Matlab has linear indexing as a column of elements, i.e, the elements are actually stored in memory as queued columns.
			So threads in X are launched in the column direction (DBT Y coord), and threads in Y are launched in the row direction (DBT X coord).

			*/
		
			threadsPerBlock.x = 32;
			threadsPerBlock.y = 32;
			threadsPerBlock.z = 1;

			blockSize.x = (nDetYMap / threadsPerBlock.x) + 1;
			blockSize.y = (nDetXMap / threadsPerBlock.y) + 1;
			blockSize.z = 1;


			mapDet2Slice_kernel << <blockSize, threadsPerBlock >> > (d_pDetmX, d_pDetmY, tubeX, rtubeY, rtubeZ, d_pDetX, d_pRdetY, d_pRdetZ, d_pObjZ, nDetXMap, nDetYMap, nz);
			//hipDeviceSynchronize();

			//mexPrintf("Map detector onto XY plane -- ThreadsX:%d ThreadsY:%d BlocksX:%d BlocksY:%d\n", threadsPerBlock.x, threadsPerBlock.y, blockSizeX, blockSizeY);

			/*
			Creates texture memory
			*/

			/*

			// Allocate CUDA array in device memory
			hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0,hipChannelFormatKindFloat);

			hipArray* d_imageArray;
			hipMallocArray(&d_imageArray, &channelDesc, nPixXMap, nPixYMap);

			// Copy to texture memory
			checkCudaErrors(hipMemcpyToArray(d_imageArray, 0, 0, d_pSlice, nPixXMap* nPixYMap * sizeof(double), hipMemcpyDeviceToDevice));

			// Specify texture
			struct hipResourceDesc resDesc;
			memset(&resDesc, 0, sizeof(resDesc));
			resDesc.resType = hipResourceTypeArray;
			resDesc.res.array.array = d_imageArray;

			// Specify texture object parameters
			struct hipTextureDesc texDesc;
			memset(&texDesc, 0, sizeof(texDesc));
			texDesc.addressMode[0] = hipAddressModeBorder;
			texDesc.addressMode[1] = hipAddressModeBorder;
			texDesc.filterMode = hipFilterModeLinear;
			texDesc.readMode = hipReadModeElementType;
			texDesc.normalizedCoords = true;

			// Create texture object
			hipTextureObject_t texObj = 0;
			hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

			// Allocate result of transformation in device memory
			double* d_sliceI;
			hipMalloc(&d_sliceI, nDetXMap*nDetYMap * sizeof(double));

			// Invoke kernel
			blockSize.x = (nDetYMap / threadsPerBlock.x) + 1;
			blockSize.y = (nDetXMap / threadsPerBlock.y) + 1;

			const double2 normFactor = make_double2(nPixX*-dx,
			nPixY*dy);

			const double2 offset = make_double2(1,
			0.5);

			bilinear_interp_kernel << <blockSize, threadsPerBlock >> > (d_sliceI, texObj, d_pDetmX, d_pDetmY, nDetXMap, nDetYMap, normFactor, offset);

			mexPrintf("Texture interpolation -- Threads:%d Blocks:%d \n", threads, blocks);

			Destroy texture object
			hipDestroyTextureObject(texObj);
			*/


			/*
			S.2. Interpolation - Liu et al (2017)
			*/

			blockSize.x = (nPixYMap / threadsPerBlock.x) + 1;
			blockSize.y = (nPixXMap / threadsPerBlock.y) + 1;
			
			bilinear_interpolation_kernel_GPU << <blockSize, threadsPerBlock >> > (d_sliceI, d_pProj, d_pObjX, d_pObjY, d_pDetmX_tmp, d_pDetmY, nPixXMap, nPixYMap, nDetXMap, nDetYMap, nDetX, nDetY, p);


			/*
			S.3. Differentiation - Eq. 24 - Liu et al (2017)
			*/

			blockSize.x = (nPixY / threadsPerBlock.x) + 1;
			blockSize.y = (nPixX / threadsPerBlock.y) + 1;

			differentiation_kernel << <blockSize, threadsPerBlock >> > (d_pVolume, d_sliceI, tubeX, rtubeY, rtubeZ, d_pObjX, d_pObjY, d_pObjZ, nPixX, nPixY, nPixXMap, nPixYMap, du, dv, dx, dy, dz, nz);

		} // Loop end slices

	} // Loop end Projections


	// Normalize volume dividing by the number of projs
	threadsPerBlock.x = 10;
	threadsPerBlock.y = 10;
	threadsPerBlock.z = 10;

	blockSize.x = (nPixY / threadsPerBlock.x) + 1;
	blockSize.y = (nPixX / threadsPerBlock.y) + 1;
	blockSize.z = (nSlices / threadsPerBlock.z) + 1;

	division_kernel << <blockSize, threadsPerBlock >> > (d_pVolume, nPixX, nPixY, nSlices, nProj2Run);

	//double* const h_var1 = (double*)mxMalloc(1 * sizeof(double));
	//hipMemcpy((void *)h_var1, (void *)d_pDetmX_tmp, 1 * sizeof(double), hipMemcpyDeviceToHost);
	//mexPrintf("P: %d Nz: %d Var:%f \n",p,nz,h_var1[0]);

	//nDetYMap
	//double* const h_pPixmX = (double*)mxMalloc(nDetXMap*nDetYMap * sizeof(double));
	//double* const h_pPixmY = (double*)mxMalloc(nDetYMap * sizeof(double));
	//hipMemcpy((void *)h_pPixmX, (void *)d_pDetmX, nDetXMap*nDetYMap * sizeof(double), hipMemcpyDeviceToHost);
	//hipMemcpy((void *)h_pPixmY, (void *)d_pDetmY, nDetYMap * sizeof(double), hipMemcpyDeviceToHost);

	//for (int u = 0; u < nDetXMap; u++)
	//	for (int v = 0; v < nDetYMap; v++) {
	//		h_pProj[(0 * nDetYMap*nDetXMap) + (u*nDetYMap) + v] = h_pPixmX[(u*nDetYMap) + v];
	//		h_pProj[(1 * nDetYMap*nDetXMap) + (u*nDetYMap) + v] = h_pPixmY[v];
	//	}

	// d_sliceI
	//double* const h_var = (double*)mxMalloc(nPixXMap*nPixYMap * sizeof(double));
	//hipMemcpy((void *)h_var, (void *)d_sliceI, nPixXMap*nPixYMap * sizeof(double), hipMemcpyDeviceToHost);

	//for (int u = 0; u < nPixXMap; u++)
	//	for (int v = 0; v < nPixYMap; v++) {
	//		h_pVolume[(0 * nPixYMap*nPixXMap) + (u*nPixYMap) + v] = h_var[(u*nPixYMap) + v];
	//	}
	

	// d_pProj
	//hipMemcpy((void *)h_pVolume, (void *)d_pProj, nProj*nDetXMap*nDetYMap * sizeof(double), hipMemcpyDeviceToHost);

	// d_pVolume
	hipMemcpy((void *)h_pVolume, (void *)d_pVolume, nSlices* nPixX * nPixY * sizeof(double), hipMemcpyDeviceToHost);
	cudaCheckErrors("hipMemcpy Final");
	

	hipFree(d_pProj);
	hipFree(d_sliceI);
	hipFree(d_pVolume);
	hipFree(d_pTubeAngle);
	hipFree(d_pDetAngle);
	hipFree(d_pDetX);
	hipFree(d_pDetY);
	hipFree(d_pDetZ);
	hipFree(d_pObjX);
	hipFree(d_pObjY);
	hipFree(d_pObjZ);
	hipFree(d_pDetmY);
	hipFree(d_pDetmX);
	hipFree(d_pRdetY);
	hipFree(d_pRdetZ);
	
	cudaCheckErrors("hipFree Final");
	
	sdkStopTimer(&timer);
	mexPrintf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
	sdkDeleteTimer(&timer);

	hipDeviceReset();

	return;

}