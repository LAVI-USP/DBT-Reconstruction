#include "hip/hip_runtime.h"
/*
%% Author: Rodrigo de Barros Vimieiro
% Date: March, 2019
% rodrigo.vimieiro@gmail.com
% =========================================================================
%{
% -------------------------------------------------------------------------
%                 projectionDDb_mex_CUDA(data3d,param)
% -------------------------------------------------------------------------
%     DESCRIPTION:
%     This function calculates the volume projection based on the
%     Branchless Distance-Driven principle.
%     The geometry is for DBT with half cone-beam. All parameters are set
%     in "ParameterSettings" code.
%
%     INPUT:
%
%     - data3d = 3D volume for projection
%     - param = Parameter of all geometry
%	  - nProj = projection number to be projected
%
%     OUTPUT:
%
%     - proj = projections for each angle.
%
%     Reference:
%     - Branchless Distance Driven Projection and Backprojection,
%     Samit Basu and Bruno De Man (2006)
%     - GPU Acceleration of Branchless Distance Driven Projection and
%     Backprojection, Liu et al (2016)
%     - GPU-Based Branchless Distance-Driven Projection and Backprojection,
%     Liu et al (2017)
%     - A GPU Implementation of Distance-Driven Computed Tomography,
%     Ryan D. Wagner (2017)
%     ---------------------------------------------------------------------
%     Copyright (C) <2019>  <Rodrigo de Barros Vimieiro>
%
%     This program is free software: you can redistribute it and/or modify
%     it under the terms of the GNU General Public License as published by
%     the Free Software Foundation, either version 3 of the License, or
%     (at your option) any later version.
%
%     This program is distributed in the hope that it will be useful,
%     but WITHOUT ANY WARRANTY; without even the implied warranty of
%     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
%     GNU General Public License for more details.
%
%     You should have received a copy of the GNU General Public License
%     along with this program.  If not, see <http://www.gnu.org/licenses/>.
%}
% =========================================================================
%% 3-D Projection Branchless Distance Driven Code
*/

#include "projectionDDb_mex_cuda.h"

/****************************************************************************
*							CUDA Kernels									*
****************************************************************************/
__global__ void pad_volume_kernel(double* d_img, const int nPixXMap, const int nPixYMap, const int nElem, unsigned int nz) {

	const int threadGId = blockIdx.x * blockDim.x + threadIdx.x;

	// Make sure we don't try and access memory outside
	// by having any threads mapped there return early
	if (threadGId >= nElem)
		return;
	
	d_img[(nz*nPixYMap *nPixXMap) + (threadGId*nPixYMap)] = 0;

	return;
}

__global__ void map_boudaries_kernel(double* d_pBound, const int nElem, const double valueLeftBound, const double sizeElem, const double offset) {

	const int threadGId = blockIdx.x * blockDim.x + threadIdx.x;

	// Make sure we don't try and access memory outside
	// by having any threads mapped there return early
	if (threadGId >= nElem)
		return;

	d_pBound[threadGId] = (threadGId - valueLeftBound) * sizeElem + offset;

	return;
}

__global__ void rot_detector_kernel(double* d_pRdetY, double* d_pRdetZ, double* d_pYcoord, double* d_pZcoord, const double yOffset, const double zOffset, 
									const double phi, const int nElem) {

	const int threadGId = blockIdx.x * blockDim.x + threadIdx.x;

	// Make sure we don't try and access memory outside
	// by having any threads mapped there return early
	if (threadGId >= nElem)
		return;

	// cos and sin are in measured in radians.

	d_pRdetY[threadGId] = ((d_pYcoord[threadGId] - yOffset)* cos(phi) - (d_pZcoord[threadGId] - zOffset)* sin(phi)) + yOffset;
	d_pRdetZ[threadGId] = ((d_pYcoord[threadGId] - yOffset)* sin(phi) + (d_pZcoord[threadGId] - zOffset)* cos(phi)) + zOffset;

	return;

}

__global__ void mapDet2Slice_kernel(double* const pXmapp, double* const pYmapp, double tubeX, double tubeY, double tubeZ, double* const pXcoord, 
									double* const pYcoord, double* const pZcoord, double* const pZSlicecoord, const int nDetXMap, const int nDetYMap, const unsigned int nz) {

	/*

	Note: Matlab has linear indexing as a column of elements, i.e, the elements are actually stored in memory as queued columns. 
	So threads in X are launched in the column direction (DBT Y coord), and threads in Y are launched in the row direction (DBT X coord).

	*/

	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
									     blockIdx.y * blockDim.y + threadIdx.y);

	const int thread_1D_pos = thread_2D_pos.y * nDetYMap + thread_2D_pos.x;

	// Make sure we don't try and access memory outside the detector
	// by having any threads mapped there return early
	if (thread_2D_pos.x >= nDetYMap || thread_2D_pos.y >= nDetXMap)
		return;

	pXmapp[thread_1D_pos] = ((pXcoord[thread_2D_pos.y] - tubeX)*(pZSlicecoord[nz] - pZcoord[thread_2D_pos.x]) - (pXcoord[thread_2D_pos.y]*tubeZ) + (pXcoord[thread_2D_pos.y]*pZcoord[thread_2D_pos.x]))/(-tubeZ + pZcoord[thread_2D_pos.x]);

	if (thread_2D_pos.y == 0)
		pYmapp[thread_2D_pos.x] = ((pYcoord[thread_2D_pos.x] - tubeY)*(pZSlicecoord[nz] - pZcoord[thread_2D_pos.x]) - (pYcoord[thread_2D_pos.x]*tubeZ) + (pYcoord[thread_2D_pos.x]*pZcoord[thread_2D_pos.x]))/(-tubeZ + pZcoord[thread_2D_pos.x]);

	return;
}

__global__ void img_integration_kernel(double* d_img, const int nPixX, const int nPixY, bool direction, unsigned int offsetX, unsigned int offsetY, unsigned int nSlices) {

	/*

	Integration of 2D slices over the whole volume
	
	(S.1.Integration. - Liu et al(2017))

	** Perfom an inclusive scan **

	*/

	const int3 memory_2D_pos = make_int3(blockIdx.x * blockDim.x + threadIdx.x + offsetX,
										 blockIdx.y * blockDim.y + threadIdx.y + offsetY,
										 blockIdx.z * blockDim.z + threadIdx.z);

	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
										 blockIdx.y * blockDim.y + threadIdx.y);


	// Make sure we don't try and access memory outside the detector
	// by having any threads mapped there return early
	if (memory_2D_pos.x >= nPixY || memory_2D_pos.y >= nPixX || memory_2D_pos.z >= nSlices)
		return;


	if (direction == integrateXcoord){

		for (int s = 1; s <= blockDim.y; s *= 2) {

			int spot = thread_2D_pos.y - s;

			double val = 0;

			if (spot >= 0) {
				val = d_img[(memory_2D_pos.z*nPixY*nPixX) + (offsetY + spot) * nPixY + memory_2D_pos.x];
			}
			__syncthreads();

			if (spot >= 0) {
				d_img[(memory_2D_pos.z*nPixY*nPixX) + (memory_2D_pos.y * nPixY) + memory_2D_pos.x] += val;
			}
			__syncthreads();
		}
	}
	else
	{

		for (int s = 1; s <= blockDim.x; s *= 2) {

			int spot = thread_2D_pos.x - s;

			double val = 0;

			if (spot >= 0) {
				val = d_img[(memory_2D_pos.z*nPixY*nPixX) + memory_2D_pos.y * nPixY + spot + offsetX];
			}
			__syncthreads();

			if (spot >= 0) {
				d_img[(memory_2D_pos.z*nPixY*nPixX) + (memory_2D_pos.y * nPixY) + memory_2D_pos.x] += val;
			}
			__syncthreads();
		}

	}
	return;
}

//__global__ void bilinear_interp_kernel(double* d_interpData, hipTextureObject_t texObj, double* const d_pDetmX, double* const d_pDetmY, const int nDetXMap, const int nDetYMap, const double2 normFactor, const double2 offset)
//{
//
//	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
//										 blockIdx.y * blockDim.y + threadIdx.y);
//
//	// Make sure we don't try and access memory outside the detector
//	// by having any threads mapped there return early
//	if (thread_2D_pos.x >= nDetYMap || thread_2D_pos.y >= nDetXMap)
//		return;
//
//	// Normalize coordinates from 0-1 (coords on texture memory)
//	double xQuery = d_pDetmX[thread_2D_pos.y * nDetYMap + thread_2D_pos.x] / normFactor.x + offset.x + (-.5f * (1 / normFactor.x));
//	double yQuery = d_pDetmY[thread_2D_pos.x] / normFactor.y + offset.y + (.5f * (1 / normFactor.y));
//
//	// Read from texture and write to global memory
//	d_interpData[thread_2D_pos.y * nDetYMap + thread_2D_pos.x] = tex2D<double>(texObj, xQuery, yQuery);
//
//	return;
//}

__global__ void bilinear_interpolation_kernel_GPU(double* d_projI, double* d_pVolume, double* d_pDetmX, double* d_pDetmY, const int nDetXMap, const int nDetYMap,
												  const int nPixXMap, const int nPixYMap, const int nPixX, const int nPixY, const double pixelSize, const unsigned int nz) {

	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
										 blockIdx.y * blockDim.y + threadIdx.y);

	// Make sure we don't try and access memory outside the detector
	// by having any threads mapped there return early
	if (thread_2D_pos.x >= nDetYMap || thread_2D_pos.y >= nDetXMap)
		return;

	/*

	S.2. Interpolation - Liu et al (2017)

	Reference:
	- https://en.wikipedia.org/wiki/Bilinear_interpolation
	- https://stackoverflow.com/questions/21128731/bilinear-interpolation-in-c-c-and-cuda

	Note: *** We are using the Unit Square equation ***

	alpha = X - X1
	1-alpha = X2 - X

	beta = Y - Y1
	1-beta = Y2 - Y

	----> Xcoord (thread_2D_pos.y)
	|							0___________
	v							|_d00_|_d10_|
	Ycoord (thread_2D_pos.x)	|_d01_|_d11_|

	*/

	// Adjust the mapped coordinates to cross the range of (0-nPixX).*dx 
	//  Divide by pixelSize to get a unitary pixel size
	const double xNormData = nPixX - d_pDetmX[thread_2D_pos.y * nDetYMap + thread_2D_pos.x] / pixelSize;
	const signed int    xData = floor(xNormData);
	const double  alpha = xNormData - xData;

	// Adjust the mapped coordinates to cross the range of (0-nPixY).*dy  
	//  Divide by pixelSize to get a unitary pixel size
	const double yNormData = (nPixY / 2.0) + (d_pDetmY[thread_2D_pos.x] / pixelSize);
	const signed int    yData = floor(yNormData);
	const double  beta = yNormData - yData;

	double d00, d01, d10, d11;
	if (((xNormData) >= 0) && ((xNormData) <= nPixX) && ((yNormData) >= 0) && ((yNormData) <= nPixY))	d00 = d_pVolume[(nz*nPixYMap*nPixXMap) + (xData*nPixYMap + yData)];						else    d00 = 0.0;
	if (((xData + 1) > 0) && ((xData + 1) <= nPixX) && ((yNormData) >= 0) && ((yNormData) <= nPixY))	d10 = d_pVolume[(nz*nPixYMap*nPixXMap) + ((xData + 1)*nPixYMap + yData)]; 				else    d10 = 0.0;
	if (((xNormData) >= 0) && ((xNormData) <= nPixX) && ((yData + 1) > 0) && ((yData + 1) <= nPixY))	d01 = d_pVolume[(nz*nPixYMap*nPixXMap) + (xData*nPixYMap + yData + 1)]; 				else    d01 = 0.0;
	if (((xData + 1) > 0) && ((xData + 1) <= nPixX) && ((yData + 1) > 0) && ((yData + 1) <= nPixY))		d11 = d_pVolume[(nz*nPixYMap*nPixXMap) + ((xData + 1)*nPixYMap + yData + 1)];			else    d11 = 0.0;

	double result_temp1 = alpha * d10 + (-d00 * alpha + d00);
	double result_temp2 = alpha * d11 + (-d01 * alpha + d01);

	d_projI[thread_2D_pos.y * nDetYMap + thread_2D_pos.x] = beta * result_temp2 + (-result_temp1 * beta + result_temp1);

}

__global__ void differentiation_kernel(double* d_pProj, double* d_projI, double* const d_pDetmX, double* const d_pDetmY, double tubeX, double rtubeY, double rtubeZ, 
									   double* const d_pDetX, double* const d_pRdetY, double* const d_pRdetZ, const int nDetX, const int nDetY, const int nDetXMap, 
									   const int nDetYMap, const double du, const double dv, const double dx, const double dy,  const double dz, const unsigned int p) {

	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
										 blockIdx.y * blockDim.y + threadIdx.y);

	const int thread_1D_pos = (nDetX*nDetY*p) + (thread_2D_pos.y * nDetY) + thread_2D_pos.x;

	// Make sure we don't try and access memory outside the detector
	// by having any threads mapped there return early
	if (thread_2D_pos.x >= nDetY || thread_2D_pos.y >= nDetX)
		return;

	/*

	S.3. Differentiation - Eq. 24 - Liu et al (2017)
	
	Detector integral projection
	 ___________
	|_A_|_B_|___|
	|_C_|_D_|___|
	|___|___|___| 


	(thread_2D_pos.x,thread_2D_pos.y)
	________________
	|_A_|__B__|_____|	
	|_C_|(0,0)|(0,1)|
	|___|(1,0)|(1,1)|

	Threads are lauched from D up to nDetX (thread_2D_pos.y) and nDetY (thread_2D_pos.x)
	i.e., they are running on the detector image. Thread (0,0) is on D.

	Coordinates on intergal projection:

	A = thread_2D_pos.y * nDetYMap + thread_2D_pos.x
	B = ((thread_2D_pos.y+1) * nDetYMap) + thread_2D_pos.x
	C = thread_2D_pos.y * nDetYMap + thread_2D_pos.x + 1
	D = ((thread_2D_pos.y+1) * nDetYMap) + thread_2D_pos.x + 1
	
	*/

	unsigned int coordA = thread_2D_pos.y * nDetYMap + thread_2D_pos.x;
	unsigned int coordB = ((thread_2D_pos.y + 1) * nDetYMap) + thread_2D_pos.x;
	unsigned int coordC = coordA + 1;
	unsigned int coordD = coordB + 1;


	// Detector X coord overlap calculation
	double detMapSizeX = d_pDetmX[coordA] - d_pDetmX[coordB];

	// Detector Y coord overlap calculation
	double detMapSizeY = d_pDetmY[thread_2D_pos.x+1] - d_pDetmY[thread_2D_pos.x];

	// x - ray angle in X coord
	double gamma = atan((d_pDetX[thread_2D_pos.y] + (du / 2.0) - tubeX) / rtubeZ);

	// x - ray angle in Y coord
	double alpha = atan((d_pRdetY[thread_2D_pos.x] + (dv / 2.0) - rtubeY) / rtubeZ);


	double dA, dB, dC, dD;

	dA = d_projI[coordA];
	dB = d_projI[coordB];
	dC = d_projI[coordC];
	dD = d_projI[coordD];

	// Treat border of interpolated integral detector
	if (dC == 0 && dD == 0) {
		dC = dA;
		dD = dB;
	}


	// S.3.Differentiation - Eq. 24 - Liu et al(2017)
	d_pProj[thread_1D_pos] += ((dD - dC - dB + dA)*(dx*dy*dz / (cos(alpha)*cos(gamma)*detMapSizeX*detMapSizeY)));

	return;
}



/****************************************************************************
* function: projectionDDb() - CUDA projection Branchless Distance Driven.	*
****************************************************************************/

void projectionDDb(double* const h_pProj,
	double* const h_pVolume,
	double* const h_pTubeAngle,
	double* const h_pDetAngle,
	const double idXProj,
	const unsigned int nProj,
	const unsigned int nPixX,
	const unsigned int nPixY,
	const unsigned int nSlices,
	const unsigned int nDetX,
	const unsigned int nDetY,
	const double dx,
	const double dy,
	const double dz,
	const double du,
	const double dv,
	const double DSD,
	const double DDR,
	const double DAG)
{

	// Unique GPU
	int devID = 0;

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, devID);
	cudaCheckErrors("hipGetDeviceProperties");

	const unsigned int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;


	mexPrintf("GPU Device %d: \"%s\" with compute capability %d.%d has %d Multi-Processors and %zu bytes of global memory\n\n", devID,
		deviceProp.name, deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount, deviceProp.totalGlobalMem);


	// Create timmer variables and start it
	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	//hipStream_t stream1;
	//hipStreamCreate(&stream1);

	dim3 threadsPerBlock(1, 1, 1);
	dim3 blockSize(1, 1, 1);


	// Number of mapped detector and pixels
	const int nDetXMap = nDetX + 1;
	const int nDetYMap = nDetY + 1;
	const int nPixXMap = nPixX + 1;
	const int nPixYMap = nPixY + 1;

	
	// Convention: h_ variables live on host
	// Convention: d_ variables live on device (GPU global mem)
	double* d_pProj;
	double* d_projI;
	double* d_pVolume;
	double* d_pTubeAngle;
	double* d_pDetAngle;


	// Allocate global memory on the device, place result in "d_----"
	hipMalloc((void **)&d_pProj, nDetX*nDetY*nProj * sizeof(double));
	hipMalloc((void **)&d_projI, nDetXMap*nDetYMap * sizeof(double));
	hipMalloc((void **)&d_pVolume, nPixXMap*nPixYMap*nSlices * sizeof(double));
	hipMalloc((void **)&d_pTubeAngle, nProj * sizeof(double));
	hipMalloc((void **)&d_pDetAngle, nProj * sizeof(double));

	cudaCheckErrors("hipMalloc Initial");

	// Copy data from host memory "h_----" to device memory "d_----"
	hipMemcpy((void *)d_pTubeAngle, (void *)h_pTubeAngle, nProj * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy((void *)d_pDetAngle, (void *)h_pDetAngle, nProj * sizeof(double), hipMemcpyHostToDevice);

	cudaCheckErrors("hipMemcpy Angles");


	/*
		Copy volume data from host memory "h_----" to device memory "d_----" padding with zeros for image integation
	*/


	// Initialize first column and row with zeros
	double* h_pVolume_tmp;
	double* d_pVolume_tmp;

	threadsPerBlock.x = maxThreadsPerBlock;
	blockSize.x = (nPixXMap / maxThreadsPerBlock) + 1;

	for (int nz = 0; nz < nSlices; nz++) {

		// Pad on X coord direction
		pad_volume_kernel << <blockSize, threadsPerBlock >> > (d_pVolume, nPixXMap, nPixYMap, nPixXMap, nz);

		// Pad on Y coord direction
		d_pVolume_tmp = d_pVolume + (nPixXMap*nPixYMap*nz) + 1;
		hipMemset(d_pVolume_tmp, 0, nPixY * sizeof(double));
		cudaCheckErrors("hipMemset Padding Volume");
	}


	// Copy volume data from host memory
	for (unsigned int nz = 0; nz < nSlices; nz++)
		for (unsigned int c = 0; c < nPixX; c++) {

			h_pVolume_tmp = h_pVolume + (c *nPixY) + (nPixX*nPixY*nz);
			d_pVolume_tmp = d_pVolume + (((c+1) *nPixYMap) + 1) + (nPixXMap*nPixYMap*nz);

			hipMemcpy((void *)d_pVolume_tmp, (void *)h_pVolume_tmp, nPixY * sizeof(double), hipMemcpyHostToDevice);
			cudaCheckErrors("hipMemset Volume");
		}

	

	// Pointer for projections coordinates
	double* d_pDetX;
	double* d_pDetY;
	double* d_pDetZ;
	double* d_pObjX;
	double* d_pObjY;
	double* d_pObjZ;

	// Allocate global memory on the device for projections coordinates
	hipMalloc((void **)&d_pDetX, nDetXMap * sizeof(double));
	hipMalloc((void **)&d_pDetY, nDetYMap * sizeof(double));
	hipMalloc((void **)&d_pDetZ, nDetYMap * sizeof(double));
	hipMalloc((void **)&d_pObjX, nPixXMap * sizeof(double));
	hipMalloc((void **)&d_pObjY, nPixYMap * sizeof(double));
	hipMalloc((void **)&d_pObjZ, nSlices * sizeof(double));

	cudaCheckErrors("hipMalloc Coordinates");



	// Pointer for mapped coordinates
	double* d_pDetmY;
	double* d_pDetmX;


	// Allocate global memory on the device for mapped coordinates
	hipMalloc((void **)&d_pDetmY, nDetYMap * sizeof(double));
	hipMalloc((void **)&d_pDetmX, nDetYMap * nDetXMap * sizeof(double));

	cudaCheckErrors("hipMalloc Map-Coordinates");


	// Pointer for rotated detector coords
	double* d_pRdetY;
	double* d_pRdetZ;

	// Allocate global memory on the device for for rotated detector coords
	hipMalloc((void **)&d_pRdetY, nDetYMap * sizeof(double));
	hipMalloc((void **)&d_pRdetZ, nDetYMap * sizeof(double));

	cudaCheckErrors("hipMalloc Rot-Coordinates");

	// Generate detector and object boudaries

	threadsPerBlock.x = maxThreadsPerBlock;

	blockSize.x = (nDetX / maxThreadsPerBlock) + 1;

	map_boudaries_kernel << <blockSize, threadsPerBlock >> > (d_pDetX, nDetXMap, (double)nDetX, -du, 0.0);

	blockSize.x = (nDetY / maxThreadsPerBlock) + 1;

	map_boudaries_kernel << <blockSize, threadsPerBlock >> > (d_pDetY, nDetYMap, nDetY / 2.0, dv, 0.0);

	blockSize.x = (nPixX / maxThreadsPerBlock) + 1;

	map_boudaries_kernel << <blockSize, threadsPerBlock >> > (d_pObjX, nPixXMap, (double)nPixX, -dx, 0.0);

	blockSize.x = (nPixY / maxThreadsPerBlock) + 1;

	map_boudaries_kernel << <blockSize, threadsPerBlock >> > (d_pObjY, nPixYMap, nPixY / 2.0, dy, 0.0);

	blockSize.x = (nSlices / maxThreadsPerBlock) + 1;

	map_boudaries_kernel << <blockSize, threadsPerBlock >> > (d_pObjZ, nSlices, 0.0, dz, DAG + (dz / 2.0));


	//mexPrintf("Map Boundaries -- Threads:%d Blocks:%d \n", threads, blocks);


	// Initiate variables value with 0
	hipMemset(d_pDetZ, 0, nDetYMap * sizeof(double));
	hipMemset(d_pProj, 0, nDetX * nDetY * nProj * sizeof(double));

	cudaCheckErrors("hipMemset Zeros");

	//hipDeviceSynchronize();

	// X - ray tube initial position
	double tubeX = 0;
	double tubeY = 0;
	double tubeZ = DSD;

	// Iso - center position
	double isoY = 0;
	double isoZ = DDR;


	// Integration of 2D slices over the whole volume
	// (S.1.Integration. - Liu et al(2017))
		
	// Naive integration o the X coord
	threadsPerBlock.x = 10;
	threadsPerBlock.y = 10;
	threadsPerBlock.z = 10;

	blockSize.x = (unsigned int) ceil(((double)nPixYMap / (threadsPerBlock.x-1)));
	blockSize.y = 1;
	blockSize.z = (unsigned int) ceil((double)nSlices / threadsPerBlock.z);

	//mexPrintf("Divisao: %f \n", (double)nPixY / (threadsPerBlock.x - 1));
	//mexPrintf("Divisao ceil: %d \n", (unsigned int)ceil((double)nPixY / (threadsPerBlock.x - 1)));

	for (int k = 0; k < ceil((double)nPixXMap / (threadsPerBlock.x-1)); k++) {

		img_integration_kernel << <blockSize, threadsPerBlock >> > (d_pVolume, nPixXMap, nPixYMap, integrateXcoord, 0, k * 9, nSlices);

		//mexPrintf("integration kernel -- threadsX:%d blocksX:%d threadsY:%d blocksY:%d \n", threadsPerBlock.x, blockSize.x, threadsPerBlock.y, blockSize.y);

	}

	// Naive integration o the Y coord
	threadsPerBlock.x = 10;
	threadsPerBlock.y = 10;
	threadsPerBlock.z = 10;

	blockSize.x = 1;
	blockSize.y = (unsigned int) ceil((double)nPixXMap / (threadsPerBlock.y-1));
	blockSize.z = (unsigned int) ceil((double)nSlices / threadsPerBlock.z);


	for (int k = 0; k < ceil((double)nPixYMap / (threadsPerBlock.y-1)); k++) {

		img_integration_kernel << <blockSize, threadsPerBlock >> > (d_pVolume, nPixXMap, nPixYMap, integrateYcoord, k * 9, 0, nSlices);

		//mexPrintf("integration kernel -- threadsX:%d blocksX:%d threadsY:%d blocksY:%d \n", threadsPerBlock.x, blockSize.x, threadsPerBlock.y, blockSize.y);

	}

	// Test if we will loop over all projs or not
	unsigned int projIni, projEnd;
	if(idXProj == -1){
		projIni = 0;
		projEnd = nProj;
	}
	else{
		projIni = (unsigned int) idXProj;
		projEnd = (unsigned int) idXProj + 1;
	}

	// For each projection
	for (unsigned int p = projIni; p < projEnd; p++) {
		
		// Get specif tube angle for the projection
		double theta = h_pTubeAngle[p] * M_PI / 180.0;

		// Get specif detector angle for the projection
		double phi = h_pDetAngle[p] * M_PI / 180.0;

		//mexPrintf("Tube angle:%f Det angle:%f\n", theta, phi);

		// Tube rotation
		double rtubeY = ((tubeY - isoY)*cos(theta) - (tubeZ - isoZ)*sin(theta)) + isoY;
		double rtubeZ = ((tubeY - isoY)*sin(theta) + (tubeZ - isoZ)*cos(theta)) + isoZ;

		//mexPrintf("R tube Y:%f R tube Z:%f\n", rtubeY, rtubeZ);

		// Detector rotation
		threadsPerBlock.x = maxThreadsPerBlock;
		threadsPerBlock.y = 1;
		threadsPerBlock.z = 1;

		blockSize.x = (nDetYMap / maxThreadsPerBlock) + 1;
		blockSize.y = 1;
		blockSize.z = 1;

		rot_detector_kernel << <blockSize, threadsPerBlock >> > (d_pRdetY, d_pRdetZ, d_pDetY, d_pDetZ, isoY, isoZ, phi, nDetYMap);

		//hipDeviceSynchronize();

		//mexPrintf("Detector rotation -- Threads:%d Blocks:%d \n", threads, blocks);


		// For each slice
		for (unsigned int nz = 0; nz < nSlices; nz++) {

			/*

			Map detector onto XY plane(Inside proj loop in case detector rotates)

			*** Note: Matlab has linear indexing as a column of elements, i.e, the elements are actually stored in memory as queued columns.
			So threads in X are launched in the column direction (DBT Y coord), and threads in Y are launched in the row direction (DBT X coord).

			*/

			threadsPerBlock.x = 32;
			threadsPerBlock.y = 32;
			threadsPerBlock.z = 1;

			blockSize.x = (nDetYMap / threadsPerBlock.x) + 1;
			blockSize.y = (nDetXMap / threadsPerBlock.y) + 1;
			blockSize.z = 1;


			mapDet2Slice_kernel << <blockSize, threadsPerBlock >> > (d_pDetmX, d_pDetmY, tubeX, rtubeY, rtubeZ, d_pDetX, d_pRdetY, d_pRdetZ, d_pObjZ, nDetXMap, nDetYMap, nz);
			//hipDeviceSynchronize();

			//mexPrintf("Map detector onto XY plane -- ThreadsX:%d ThreadsY:%d BlocksX:%d BlocksY:%d\n", threadsPerBlock.x, threadsPerBlock.y, blockSizeX, blockSizeY);
			
			/*
			Creates texture memory
			*/

			/*

			// Allocate CUDA array in device memory
			hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0,hipChannelFormatKindFloat);

			hipArray* d_imageArray;
			hipMallocArray(&d_imageArray, &channelDesc, nPixXMap, nPixYMap);
				
			// Copy to texture memory			
			checkCudaErrors(hipMemcpyToArray(d_imageArray, 0, 0, d_pSlice, nPixXMap* nPixYMap * sizeof(double), hipMemcpyDeviceToDevice));

			// Specify texture
			struct hipResourceDesc resDesc;
			memset(&resDesc, 0, sizeof(resDesc));
			resDesc.resType = hipResourceTypeArray;
			resDesc.res.array.array = d_imageArray;

			// Specify texture object parameters
			struct hipTextureDesc texDesc;
			memset(&texDesc, 0, sizeof(texDesc));
			texDesc.addressMode[0] = hipAddressModeBorder;
			texDesc.addressMode[1] = hipAddressModeBorder;
			texDesc.filterMode = hipFilterModeLinear;
			texDesc.readMode = hipReadModeElementType;
			texDesc.normalizedCoords = true;

			// Create texture object
			hipTextureObject_t texObj = 0;
			hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

			// Allocate result of transformation in device memory
			double* d_projI;
			hipMalloc(&d_projI, nDetXMap*nDetYMap * sizeof(double));

			// Invoke kernel
			blockSize.x = (nDetYMap / threadsPerBlock.x) + 1;
			blockSize.y = (nDetXMap / threadsPerBlock.y) + 1;

			const double2 normFactor = make_double2(nPixX*-dx,
												  nPixY*dy);

			const double2 offset = make_double2(1,
											  0.5);

			bilinear_interp_kernel << <blockSize, threadsPerBlock >> > (d_projI, texObj, d_pDetmX, d_pDetmY, nDetXMap, nDetYMap, normFactor, offset);

			mexPrintf("Texture interpolation -- Threads:%d Blocks:%d \n", threads, blocks);

			 Destroy texture object
			hipDestroyTextureObject(texObj);
			*/


			/*
				S.2. Interpolation - Liu et al (2017)
			*/

			blockSize.x = (nDetYMap / threadsPerBlock.x) + 1;
			blockSize.y = (nDetXMap / threadsPerBlock.y) + 1;

			bilinear_interpolation_kernel_GPU << <blockSize, threadsPerBlock >> > (d_projI, d_pVolume, d_pDetmX, d_pDetmY, nDetXMap, nDetYMap, nPixXMap, nPixYMap, nPixX, nPixY, dx, nz);


			/*
				S.3. Differentiation - Eq. 24 - Liu et al (2017)
			*/

			blockSize.x = (nDetY / threadsPerBlock.x) + 1;
			blockSize.y = (nDetX / threadsPerBlock.y) + 1;

			differentiation_kernel << <blockSize, threadsPerBlock >> > (d_pProj, d_projI, d_pDetmX, d_pDetmY, tubeX, rtubeY, rtubeZ, d_pDetX, d_pRdetY, d_pRdetZ, nDetX, nDetY, nDetXMap, nDetYMap, du, dv, dx, dy, dz, p);

		} // Loop end slices

	} // Loop end Projections


	//nDetYMap
	//double* const h_pPixmX = (double*)mxMalloc(nDetXMap*nDetYMap * sizeof(double));
	//double* const h_pPixmY = (double*)mxMalloc(nDetYMap * sizeof(double));
	//hipMemcpy((void *)h_pPixmX, (void *)d_pDetmX, nDetXMap*nDetYMap * sizeof(double), hipMemcpyDeviceToHost);
	//hipMemcpy((void *)h_pPixmY, (void *)d_pDetmY, nDetYMap * sizeof(double), hipMemcpyDeviceToHost);

	//for (int u = 0; u < nDetXMap; u++)
	//	for (int v = 0; v < nDetYMap; v++) {
	//		h_pProj[(0 * nDetYMap*nDetXMap) + (u*nDetYMap) + v] = h_pPixmX[(u*nDetYMap) + v];
	//		h_pProj[(1 * nDetYMap*nDetXMap) + (u*nDetYMap) + v] = h_pPixmY[v];
	//	}
	
	// d_projI
	//double* const h_pPixmX = (double*)mxMalloc(nDetXMap*nDetYMap * sizeof(double));
	//hipMemcpy((void *)h_pPixmX, (void *)d_projI, nDetXMap*nDetYMap * sizeof(double), hipMemcpyDeviceToHost);

	//for (int u = 0; u < nDetXMap; u++)
	//	for (int v = 0; v < nDetYMap; v++) {
	//		h_pProj[(0 * nDetYMap*nDetXMap) + (u*nDetYMap) + v] = h_pPixmX[(u*nDetYMap) + v];
	//	}
	

	// d_pVolume 
	//hipMemcpy((void *)h_pProj, (void *)d_pVolume, nSlices*nPixXMap*nPixYMap * sizeof(double), hipMemcpyDeviceToHost);

	// d_pProj
	hipMemcpy((void *)h_pProj, (void *)d_pProj, nProj*	nDetX * nDetY * sizeof(double), hipMemcpyDeviceToHost);
	cudaCheckErrors("hipMemcpy Final");


	hipFree(d_pProj);
	hipFree(d_projI);
	hipFree(d_pVolume);
	hipFree(d_pTubeAngle);
	hipFree(d_pDetAngle);
	hipFree(d_pDetX);
	hipFree(d_pDetY);
	hipFree(d_pDetZ);
	hipFree(d_pObjX);
	hipFree(d_pObjY);
	hipFree(d_pObjZ);
	hipFree(d_pDetmY);
	hipFree(d_pDetmX);
	hipFree(d_pRdetY);
	hipFree(d_pRdetZ);

	cudaCheckErrors("hipFree Final");

	sdkStopTimer(&timer);
	mexPrintf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
	sdkDeleteTimer(&timer);

	hipDeviceReset();

	return;

}